
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

void initWith(float num, float *a, int N)
{
	for (int i = 0; i < N; ++i)
		a[i] = num;
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = gridDim.x * blockDim.x;

	for (int i = index; i < N; i += stride)
		result[i] = a[i] + b[i];
}

void checkElementsAre(float target, float *array, int N)
{
	for (int i = 0; i < N; i++) {
		if (array[i] != target) {
			printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
			exit(1);
		}
	}
	printf("SUCCESS! All values added correctly.\n");
}

int main(void)
{
	const int N = 2 << 20;
	size_t size = N * sizeof(float);

	float *a;
	float *b;
	float *c;

	checkCuda( hipMallocManaged(&a, size) );
	checkCuda( hipMallocManaged(&b, size) );
	checkCuda( hipMallocManaged(&c, size) );

	initWith(3, a, N);
	initWith(4, b, N);
	initWith(0, c, N);

	const size_t threads_per_block = 32;
	const size_t block_size = (N + threads_per_block - 1) / threads_per_block; // at most one extra block

	addVectorsInto<<<block_size,threads_per_block>>>(c, a, b, N);
	checkCuda( hipGetLastError() );

	checkCuda( hipDeviceSynchronize() );
	checkElementsAre(7, c, N);

	hipFree(a);
	hipFree(b);
	hipFree(c);
}
