#include "hip/hip_runtime.h"
#include <stdio.h>

#define N (2048 * 2048) // Number of elements in each vector
#define X (2)

__global__
void saxpy(int *a, int *b, int *c)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < N; i += stride)
		c[i] = a[i]*X + b[i];
}

int main(void)
{
	int *a, *b, *c;
	size_t size = N * sizeof(int);

	int deviceId;
	hipGetDevice(&deviceId);

	int numberOfSMs;
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	hipMallocManaged(&a, size);
	hipMallocManaged(&b, size);
	hipMallocManaged(&c, size);

	for (int i = 0; i < N; ++i) {
		a[i] = 2;
		b[i] = 1;
		c[i] = 0;
	}

	hipMemPrefetchAsync(a, size, deviceId);
	hipMemPrefetchAsync(b, size, deviceId);
	hipMemPrefetchAsync(c, size, deviceId);

	size_t threads_per_block = 256;
	size_t number_of_blocks = numberOfSMs * 32;

	saxpy<<<number_of_blocks,threads_per_block>>>(a, b, c);

	hipDeviceSynchronize();

	// Print out the first and last 5 values of c for a quality check
	for (int i = 0; i < 5; ++i)
			printf("c[%d] = %d, ", i, c[i]);
	printf ("\n");
	for (int i = N-5; i < N; ++i)
			printf("c[%d] = %d, ", i, c[i]);
	printf ("\n");

	hipFree(a);
	hipFree(b);
	hipFree(c);
}
