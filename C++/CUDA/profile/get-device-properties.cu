
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
	int deviceId;
	hipGetDevice(&deviceId);

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, deviceId);

	int computeCapabilityMajor = props.major;
	int computeCapabilityMinor = props.minor;
	int multiProcessorCount = props.multiProcessorCount;
	int warpSize = props.warpSize;
	/// << other properties >

	printf(
		"Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n",
		deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize
	);
}
