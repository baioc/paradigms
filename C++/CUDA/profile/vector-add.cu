#include "hip/hip_runtime.h"
#include <stdio.h>

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */
__global__
void initWith(float num, float *a, int N)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < N; i += stride)
		a[i] = num;
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */
__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < N; i += stride)
		result[i] = a[i] + b[i];
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */
void checkElementsAre(float target, float *vector, int N)
{
	for (int i = 0; i < N; i++) {
		if (vector[i] != target) {
			printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
			exit(1);
		}
	}
	printf("Success! All values calculated correctly.\n");
}

int main(void)
{
	const int N = 2 << 24;
	size_t size = N * sizeof(float);

	float *a;
	float *b;
	float *c;

	hipMallocManaged(&a, size);
	hipMallocManaged(&b, size);
	hipMallocManaged(&c, size);

	int deviceId;
	hipGetDevice(&deviceId);

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, deviceId);

	size_t threadsPerBlock = 1024;
	size_t numberOfBlocks = props.multiProcessorCount * 32; // multiple of SM number to optimize GPU occupation

	hipMemPrefetchAsync(a, size, deviceId); // Prefetch to device.
	hipMemPrefetchAsync(b, size, deviceId);
	hipMemPrefetchAsync(c, size, deviceId);

	initWith<<<numberOfBlocks,threadsPerBlock>>>(3, a, N);
	initWith<<<numberOfBlocks,threadsPerBlock>>>(4, b, N);
	initWith<<<numberOfBlocks,threadsPerBlock>>>(0, c, N);

	addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
	hipError_t addVectorsErr = hipGetLastError();
	if (addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

	hipError_t asyncErr = hipDeviceSynchronize();
	if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

	hipMemPrefetchAsync(c, size, hipCpuDeviceId); // Prefetch to host.

	checkElementsAre(7, c, N);

	hipFree(a);
	hipFree(b);
	hipFree(c);
}
