
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < N; i += stride)
		a[i] = num;
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < N; i += stride)
		result[i] = a[i] + b[i];
}

void checkElementsAre(float target, float *vector, int N)
{
	for (int i = 0; i < N; i++) {
		if (vector[i] != target) {
			printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
			exit(1);
		}
	}
	printf("Success! All values calculated correctly.\n");
}

int main()
{
	int deviceId;
	int numberOfSMs;

	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	const int N = 2 << 24;
	size_t size = N * sizeof(float);

	float *a;
	float *b;
	float *c;
	float *h_c;

	hipMalloc(&a, size);
	hipMalloc(&b, size);
	hipMalloc(&c, size);
	hipHostMalloc(&h_c, size, hipHostMallocDefault);

	size_t threadsPerBlock;
	size_t numberOfBlocks;

	threadsPerBlock = 256;
	numberOfBlocks = 32 * numberOfSMs;

	/*
	 * Create 3 streams to run initialize the 3 data vectors in parallel.
	 */
	hipStream_t stream1, stream2, stream3;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);

	const int numberOfSegments = 4;                  // This example demonstrates slicing the work into 4 segments.
	int segmentN = N / numberOfSegments;             // A value for a segment's worth of `N` is needed.
	size_t segmentSize = size / numberOfSegments;    // A value for a segment's worth of `size` is needed.

	// For each of the 4 segments...
	for (int i = 0; i < numberOfSegments; ++i) {
		// Calculate the index where this particular segment should operate within the larger arrays.
		size_t segmentOffset = i * segmentN;

		// Create a stream for this segment's worth of copy and work.
		hipStream_t stream;
		hipStreamCreate(&stream);

		/*
		 * Give each `initWith` launch its own non-standard stream.
		 */
		initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3, a, segmentN);
		initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4, b, segmentN);
		initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, c, segmentN);

		// Execute segment's worth of work over same non-default stream as memory copy.
		addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream>>>(&c[segmentOffset], &a[segmentOffset], &b[segmentOffset], segmentN);

		// Asynchronously copy segment's worth of device memory to pinned host mem to over non-default stream.
		hipMemcpyAsync(&h_c[segmentOffset],    // Take care to access correct location in array.
		                &c[segmentOffset],      // Take care to access correct location in array.
		                segmentSize,            // Only copy a segment's worth of memory.
		                hipMemcpyDeviceToHost, // Copy direction.
		                stream);                // Provide optional argument for non-default stream.

		// `cudaStreamDestroy` will return immediately (is non-blocking), but will not actually destroy stream until
		// all stream operations are complete, so this is safe.
		hipStreamDestroy(stream);
	}

	hipDeviceSynchronize();
	checkElementsAre(7, h_c, N);

	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);

	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipHostFree(h_c);
}
